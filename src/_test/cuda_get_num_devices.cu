#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int count = 0;
    hipError_t error = hipGetDeviceCount(&count);
    printf("Return code: %d (%s)\n", error, hipGetErrorString(error));
    printf("Number of GPUs: %d\n", count);
    
    // Also try getting properties for each device
    for (int i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        error = hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s (Error: %s)\n", i, prop.name, hipGetErrorString(error));
    }
    
    return 0;
}